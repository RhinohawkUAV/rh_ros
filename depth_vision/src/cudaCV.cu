#include <stdio.h>
#include <hip/hip_runtime.h>



__global__  void person_threshold(unsigned char * pix, int cols, int count){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char p = (unsigned char)0;
	float ff;
	 if(i < count - cols ){
             int diffX =  ((int)pix[i+ cols] - (int)pix[i + cols- 1]);
             int diffY =  ((int)pix[i + cols] - (int)pix[i]);     
             p = (unsigned char)((int)sqrtf((diffX * diffX) + (diffY * diffY)));
__syncthreads();
	pix[i] = p;
}
__syncthreads();
if( i> 0 && count - cols-1){
	p = (pix[i-1] +  pix[i+1] + pix[i + cols] + pix[i+1 + cols])/12;
}
__syncthreads();
 pix[i] +=p;

if( i > 0 && i < count - cols - 1){
	p = (pix[i-1] +  pix[i+1] + pix[i + cols] + pix[i+1 + cols])/12;
}
__syncthreads();
 pix[i] +p;

if( i > 0 &&i < count - cols - 1){
	p = (pix[i-1] +  pix[i+1] + pix[i + cols] + pix[i+1 + cols])/12;
}
__syncthreads();
 pix[i] +=p;

int radius = 1;
if(i > cols + 1  && i < count - cols - 1 && i % cols != 0){	
	p = (
abs(pix[i-1-cols] - pix[i]) 
+ abs(pix[i-cols] - pix[i]) 
+ abs(pix[i + 1 - cols] - pix[i]) 
+ abs(pix[i -1] - pix[i]) + abs( pix[i + 1] -  pix[i]) 
+ abs(pix[i - 1 + cols] - pix[i])
 + abs(pix[i + cols] - pix[i]) 
+ abs(pix[i + 1 + cols] - pix[i])
)/8;
}

__syncthreads();
 pix[i] = p;
}
void  edgedetect(unsigned char * pic, int count, int cols){
     const int numthreads = count;
     const int blocks = numthreads/512;
     const int block_width =  512;
     unsigned char * data = NULL;
	hipMalloc(&data, count * sizeof(unsigned char));

	hipMemcpy(data, pic, count* sizeof(unsigned char), hipMemcpyHostToDevice);

	
        person_threshold<<<blocks, block_width >>>(data, cols, count);

	hipMemcpy(pic, data, count * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(data);
}
